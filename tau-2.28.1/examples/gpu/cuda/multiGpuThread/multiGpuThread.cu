
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <pthread.h>
 
#define ARR_SIZE    10
#define NUM_DEVICE 2
#define NUM_THR  8
 
typedef struct {
  int *arr;
  int *dev_arr;
  int *dev_result;
  int *result;
  int dev_num;
  int thr_num;
} cuda_st;
 
__global__ void kernel_fc(int *dev_arr, int *dev_result)
{
  int idx = threadIdx.x;
  printf("dev_arr[%d] = %d\n", idx, dev_arr[idx]);
  atomicAdd(dev_result, dev_arr[idx]);
}
 
void *thread_func(void* struc)
{
  cuda_st * data = (cuda_st*)struc;
  printf("thread %d func start\n", data->thr_num);
  printf("arr %d = ", data->dev_num);
  for(int i=0; i<10; i++) {
    printf("%d ", data->arr[i]);
  }
  printf("\n");
  hipSetDevice(data->dev_num);
  hipMemcpy(data->dev_arr, data->arr,  sizeof(int)*ARR_SIZE, hipMemcpyHostToDevice);
  kernel_fc<<<1,ARR_SIZE>>>(data->dev_arr, data->dev_result);
  hipMemcpy(data->result, data->dev_result, sizeof(int), hipMemcpyDeviceToHost);
  printf("thread %d func exit\n", data->thr_num);
  return NULL;
}
 
int main(void)
{
  // Make object
  cuda_st cuda[NUM_DEVICE][NUM_THR];
 
  // Make thread
  pthread_t pthread[NUM_DEVICE*NUM_THR];
 
  // Host array memory allocation
  int *arr[NUM_DEVICE];
  for(int i=0; i<NUM_DEVICE; i++) {
    arr[i] = (int*)malloc(sizeof(int)*ARR_SIZE);
  }
 
  // Fill this host array up with specified data
  for(int i=0; i<NUM_DEVICE; i++) {
    for(int j=0; j<ARR_SIZE; j++) {
      arr[i][j] = i*ARR_SIZE+j;
    }
  }
 
  // To confirm host array data
  for(int i=0; i<NUM_DEVICE; i++) {
    printf("arr[%d] = ", i);
    for(int j=0; j<ARR_SIZE; j++) {
      printf("%d ", arr[i][j]);
    }
    printf("\n");
  }
 
  // Result memory allocation
  int *result[NUM_DEVICE];
  for(int i=0; i<NUM_DEVICE; i++) {
    result[i] = (int*)malloc(sizeof(int));
    memset(result[i], 0, sizeof(int));
  }
 
  // Device array memory allocation
  int *dev_arr[NUM_DEVICE];
  for(int i=0; i<NUM_DEVICE; i++) {
    hipSetDevice(i);
    hipMalloc(&dev_arr[i], sizeof(int)*ARR_SIZE);
  }
 
  // Device result memory allocation
  int *dev_result[NUM_DEVICE];
  for(int i=0; i<NUM_DEVICE; i++) {
    hipSetDevice(i);
    hipMalloc(&dev_result[i], sizeof(int));
    hipMemset(dev_result[i], 0, sizeof(int));
  }
 
  // Connect these pointers with object
  for (int i=0; i<NUM_DEVICE; i++)
    for (int j=0; j<NUM_THR; j++) {
      cuda[i][j].arr = arr[i];
      cuda[i][j].dev_arr = dev_arr[i];
      cuda[i][j].result = result[i];
      cuda[i][j].dev_result = dev_result[i];
      cuda[i][j].dev_num = i;
      cuda[i][j].thr_num = j;
    }
 
  // Create and excute pthread
  for(int i=0; i<NUM_DEVICE; i++)
    for (int j=0; j<NUM_THR; j++) {
      pthread_create(&pthread[(i*NUM_THR)+j], NULL, thread_func, (void*)&cuda[i][j]);
    }
 
  // Join pthread
  for(int i=0; i<NUM_DEVICE*NUM_THR; i++) {
    pthread_join(pthread[i], NULL);
  }
 
  for(int i=0; i<NUM_DEVICE; i++)
    for (int j=0; j < NUM_THR; j++) {
      printf("result[%d][%d] = %d\n", i,j, (*cuda[i][j].result));
    }

  hipDeviceReset();
  return 0;
}

